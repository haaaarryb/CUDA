
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (i < N) // zero indexed so not <=
  {
    a[i] = initialValue;
  }
}

int main()
{
  int N = 1000;

  int *a;
  size_t size = N * sizeof(int);

  hipMallocManaged(&a, size);

  size_t threads_per_block = 256;

  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block; // ensure at least N threads in grid, but only one blocks worth extra

  int initialValue = 6;

  initializeElementsTo<<<number_of_blocks, threads_per_block>>>(initialValue, a, N);
  hipDeviceSynchronize();

  for (int i = 0; i < N; ++i)
  {
    if(a[i] != initialValue)
    {
      printf("FAILURE: target value: %d\t a[%d]: %d\n", initialValue, i, a[i]);
      exit(1);
    }
  }
  printf("SUCCESS!\n");

  hipFree(a);
}
