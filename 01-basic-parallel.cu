
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor firstParallel so that it can run on the GPU.
 */

__global__ void firstParallel()
{
  printf("This should be running in parallel.\n");
}

int main()
{
  firstParallel<<<1, 5>>>();
  hipDeviceSynchronize();

}
