
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__ void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */
  

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  
  //cudaError_t err;
  //err = cudaMallocManaged(&a, size);
  hipMallocManaged(&a, size);
  
  /*
  if (err != cudaSuccess)
  {
    printf("Here is the error: %s", cudaGetErrorString(err));
  }
  printf(err);
  */

  init(a, N);

  size_t threads_per_block = 1024;
  //size_t threads_per_block = 2048;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  
   // Catch errors for both the kernel launch above and any errors that occur during the asynchronous `doubleElements` kernel execution.
  
  hipError_t syncError = hipGetLastError();
  hipError_t asyncError = hipDeviceSynchronize();
  
  if (syncError != hipSuccess) printf("Here is an error: %s\n", hipGetErrorString(syncError));       // invalid configuration argument
  if (asyncError != hipSuccess) printf("Here is an error: %s\n", hipGetErrorString(asyncError));     // no error

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
