
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void loop()
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    printf("This is iteration number %d\n", i);
}

int main()
{
  loop<<<2, 5>>>();
  hipDeviceSynchronize();
}
