
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
  int deviceId;
  hipGetDevice(&deviceId);                        // `deviceId`  now points to the id of the currently active GPU.
  hipDeviceProp_t props;                            // initialise
  hipGetDeviceProperties(&props, deviceId);       // `props` now has many useful properties about the active GPU device.
  
  
  int computeCapabilityMajor = props.major;
  int computeCapabilityMinor = props.minor;
  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;

  printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
}
